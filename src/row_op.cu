#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include "equations.h"
#include "util.h"
#include "smem.h"
#include "debug.h"

namespace inplace {

namespace _3d {

namespace _132 {

template<typename F, typename T>
__global__ void compress_row_gather_op(F fn, T* data, size_t batch_size, int d1, int d2, int d3) {
    T* smem = shared_memory<T>();

    size_t l = chunk_left(blockIdx.x, gridDim.x, d2);
    size_t r = chunk_right(blockIdx.x, gridDim.x, d2);
	size_t d1d2 = (size_t)d1 * (size_t)d2;
	size_t d1d3 = (size_t)d1 * (size_t)d3;
    for (size_t lv = l; lv < r; lv += batch_size) {
        batch_size = min(batch_size, r - lv);
        size_t offset = lv * (size_t)d1;
        __syncthreads();
        for (size_t idx = threadIdx.x; idx < batch_size * d1d3; idx += blockDim.x) {
            smem[idx] = data[offset + idx];
        }
        
        __syncthreads();
        for (size_t idx = threadIdx.x; idx < batch_size * d1d3; idx += blockDim.x) {
            int u = (idx / d1d3);
            size_t i = (lv + u) % d2;
            size_t j = idx % d1;
			size_t k = idx / d1;
            fn.set_i(i);
            data[offset + idx] = smem[j + u * d1 + fn(k) * d1d2];
        }
    }
}

template<typename F, typename T>
__global__ void compress_row_scatter_op(F fn, T* data, size_t batch_size, int d1, int d2, int d3) {
    T* smem = shared_memory<T>();

    size_t l = chunk_left(blockIdx.x, gridDim.x, d2);
    size_t r = chunk_right(blockIdx.x, gridDim.x, d2);
	size_t d1d2 = (size_t)d1 * (size_t)d2;
	size_t d1d3 = (size_t)d1 * (size_t)d3;
    for (size_t lv = l; lv < r; lv += batch_size) {
        batch_size = min(batch_size, r - lv);
        //size_t offset = lv * (size_t)d1;
        __syncthreads();
        for (size_t idx = threadIdx.x; idx < batch_size * d1d3; idx += blockDim.x) {
			int u = (idx / d1d3);
            size_t i = (lv + u) % d2;
            size_t j = idx % d1;
			size_t k = idx / d1;
            fn.set_i(i);
            //smem[j + u * d1 + fn(k) * d1d2] = data[offset + idx];
			smem[j + u * d1 + fn(k) * d1d2] = data[j + i * d1 + k * d1d2];
        }
        
        __syncthreads();
        for (size_t idx = threadIdx.x; idx < batch_size * d1d3; idx += blockDim.x) {
            //data[offset + idx] = smem[idx];
			int u = (idx / d1d3);
            size_t i = (lv + u) % d2;
            size_t j = idx % d1;
			size_t k = idx / d1;
			data[j + i * d1 + k * d1d2] = smem[idx];
        }
    }
}

template<typename F, typename K, typename T>
void compress_row_launch(F fn, K kernel, T* data, int d1, int d2, int d3) {
	PRINT("Smem Compress %s\n", fn.getName().c_str());
	size_t smem_lim = shared_mem_per_block();
	size_t smem_size = smem_lim / 32;
	int n_threads = max_n_threads_per_sm() / 32;
	PRINT("\t# threads = %d\n", n_threads);
	int n_blocks = min(d2, get_num_block(kernel, n_threads, smem_size));
	PRINT("\t# blocks = %d\n", n_blocks);
	size_t batch_size = smem_size / (sizeof(T) * (size_t)d1 * (size_t)d3);
	PRINT("\tbatch size = %zu\n", batch_size);
	kernel<<<n_blocks, n_threads, smem_size>>>(fn, data, batch_size, d1, d2, d3);
}

template<typename F, typename T>
__global__ void smem_row_gather_op(F fn, T* data, int d1, int d2, int d3) {
    T* smem = shared_memory<T>();
    
	size_t d1d2 = (size_t)d1 * size_t(d2);
	size_t d1d3 = (size_t)d1 * size_t(d3);
	for (size_t i = blockIdx.x; i < d2; i += gridDim.x) {
		fn.set_i(i);
		size_t id1 = i * d1;
		__syncthreads();
		for(size_t idx = threadIdx.x; idx < d1d3; idx += blockDim.x) {
			size_t j = idx % d1;
			size_t k = idx / d1;
			smem[idx] = data[j + id1 + k * d1d2];
		}
		__syncthreads();
		for(size_t idx = threadIdx.x; idx < d1d3; idx += blockDim.x) {
			size_t j = idx % d1;
			size_t k = idx / d1;
			data[j + id1 + k * d1d2] = smem[j + fn(k) * d1];
		}
	}
}

template<typename F, typename T>
__global__ void smem_row_scatter_op(F fn, T* data, int d1, int d2, int d3) {
    T* smem = shared_memory<T>();
    
	size_t d1d2 = (size_t)d1 * size_t(d2);
	size_t d1d3 = (size_t)d1 * size_t(d3);
	for (size_t i = blockIdx.x; i < d2; i += gridDim.x) {
		fn.set_i(i);
		size_t id1 = i * d1;
		__syncthreads();
		for(size_t idx = threadIdx.x; idx < d1d3; idx += blockDim.x) {
			size_t j = idx % d1;
			size_t k = idx / d1;
			smem[j + fn(k) * d1] = data[j + id1 + k * d1d2];
		}
		__syncthreads();
		for(size_t idx = threadIdx.x; idx < d1d3; idx += blockDim.x) {
			size_t j = idx % d1;
			size_t k = idx / d1;
			data[j + id1 + k * d1d2] = smem[idx];
		}
	}
}

template<typename F, typename K, typename T>
void smem_row_launch(F fn, K kernel, T* data, int d1, int d2, int d3) {
	PRINT("Smem %s\n", fn.getName().c_str());
	size_t smem_size = sizeof(T) * (size_t)d1 * (size_t)d3;
	int n_threads = get_num_thread(d1 * d3);
	PRINT("\t# threads = %d\n", n_threads);
	int n_blocks = min(d2, get_num_block(kernel, n_threads, smem_size));
	kernel<<<n_blocks, n_threads, smem_size>>>(fn, data, d1, d2, d3);
}

template<typename F, typename T>
__global__ void gmem_row_gather_op(F fn, T* data, T* tmp, int d1, int d2, int d3) {
    namespace cg = cooperative_groups;
    cg::grid_group g = cg::this_grid();

	size_t d1d2 = (size_t)d1 * size_t(d2);
	size_t d1d3 = (size_t)d1 * size_t(d3);
    size_t global_id = threadIdx.x + blockIdx.x * blockDim.x;
    size_t grid_size = gridDim.x * blockDim.x;
	for (size_t i = 0; i < d2; i++) {
		fn.set_i(i);
		size_t id1 = i * d1;
		g.sync();
		for (size_t idx = global_id; idx < d1d3; idx += grid_size) {
			size_t j = idx % d1;
			size_t k = idx / d1;
			tmp[idx] = data[j + id1 + fn(k) * d1d2];
		}
		g.sync();
		for (size_t idx = global_id; idx < d1d3; idx += grid_size) {
			size_t j = idx % d1;
			size_t k = idx / d1;
			data[j + id1 + k * d1d2] = tmp[idx];
		}
	}
}

template<typename F, typename T>
__global__ void gmem_row_scatter_op(F fn, T* data, T* tmp, int d1, int d2, int d3) {
    namespace cg = cooperative_groups;
    cg::grid_group g = cg::this_grid();

	size_t d1d2 = (size_t)d1 * size_t(d2);
	size_t d1d3 = (size_t)d1 * size_t(d3);
    size_t global_id = threadIdx.x + blockIdx.x * blockDim.x;
    size_t grid_size = gridDim.x * blockDim.x;
	for (size_t i = 0; i < d2; i++) {
		fn.set_i(i);
		size_t id1 = i * d1;
		g.sync();
		for (size_t idx = global_id; idx < d1d3; idx += grid_size) {
			size_t j = idx % d1;
			size_t k = idx / d1;
			tmp[j + fn(k) * d1] = data[j + id1 + k * d1d2];
		}
		g.sync();
		for (size_t idx = global_id; idx < d1d3; idx += grid_size) {
			size_t j = idx % d1;
			size_t k = idx / d1;
			data[j + id1 + k * d1d2] = tmp[idx];
		}
	}
}

template<typename F, typename K, typename T>
void gmem_row_launch(F fn, K kernel, T* data, int d1, int d2, int d3) {
	PRINT("Gmem %s\n", fn.getName().c_str());
	T* tmp;
	size_t tmp_size = sizeof(T) * d1 * d3;
	CudaSafeCall( hipMallocManaged(&tmp, tmp_size) );
	prefetch(tmp, tmp_size);
	int n_threads = 1024;
	int n_blocks = get_num_block(kernel, n_threads, 0);
	PRINT("\t# blocks = %d\n", n_blocks);
	void *kernelArgs[] = {
		(void *)&fn, (void *)&data, (void *)&tmp, (void *)&d1, (void *)&d2, (void *)&d3
	};
	CudaSafeCall( hipLaunchCooperativeKernel((void *)kernel,
										  n_blocks, n_threads, kernelArgs) );
	CudaSafeCall( hipFree(tmp) );
}

template<typename F, typename T>
void row_gather_op(F fn, T* data, int d1, int d2, int d3) {
	size_t smem_lim = shared_mem_per_block();
	/*if (2 * d1 * d3 * sizeof(T) <= smem_lim / 32) {
		compress_row_launch(fn, compress_row_gather_op<F, T>, data, d1, d2, d3);
    }
    else*/ if (sizeof(T) * d1 * d3 <= smem_lim) {
		smem_row_launch(fn, smem_row_gather_op<F, T>, data, d1, d2, d3);
    }
	else {
        gmem_row_launch(fn, gmem_row_gather_op<F, T>, data, d1, d2, d3);
    }
}

template<typename F, typename T>
void row_scatter_op(F fn, T* data, int d1, int d2, int d3) {
	size_t smem_lim = shared_mem_per_block();
	/*if (2 * d1 * d3 * sizeof(T) <= smem_lim / 32) {
		compress_row_launch(fn, compress_row_scatter_op<F, T>, data, d1, d2, d3);
    }
    else*/ if (sizeof(T) * d1 * d3 <= smem_lim) {
		smem_row_launch(fn, smem_row_scatter_op<F, T>, data, d1, d2, d3);
    }
	else {
        gmem_row_launch(fn, gmem_row_scatter_op<F, T>, data, d1, d2, d3);
    }
}

template void row_gather_op(_2d::c2r::row_shuffle, float*, int, int, int);
template void row_gather_op(_2d::c2r::row_shuffle, double*, int, int, int);

template void row_gather_op(_2d::r2c::row_shuffle, float*, int, int, int);
template void row_gather_op(_2d::r2c::row_shuffle, double*, int, int, int);

template void row_scatter_op(_2d::r2c::row_scatter_shuffle, float*, int, int, int);
template void row_scatter_op(_2d::r2c::row_scatter_shuffle, double*, int, int, int);

}

namespace _213 {

template<typename F, typename T>
__global__ void compress_row_gather_op(F fn, T* data, size_t batch_size, int d1, int d2, int d3) {
    T* smem = shared_memory<T>();

	size_t d2d3 = (size_t)d2 * (size_t)d3;
    size_t l = chunk_left(blockIdx.x, gridDim.x, d2d3);
    size_t r = chunk_right(blockIdx.x, gridDim.x, d2d3);
    for (size_t lv = l; lv < r; lv += batch_size) {
        batch_size = min(batch_size, r - lv);
        size_t offset = lv * (size_t)d1;
        __syncthreads();
        for (size_t idx = threadIdx.x; idx < batch_size * d1; idx += blockDim.x) {
            smem[idx] = data[offset + idx];
        }
        
        __syncthreads();
        for (size_t idx = threadIdx.x; idx < batch_size * d1; idx += blockDim.x) {
            int u = (idx / d1);
            size_t i = (lv + u) % d2;
            size_t j = idx % d1;
            fn.set_i(i);
            data[offset + idx] = smem[fn(j) + u * d1];
        }
    }
}

template<typename F, typename T>
__global__ void compress_row_scatter_op(F fn, T* data, size_t batch_size, int d1, int d2, int d3) {
    T* smem = shared_memory<T>();

    size_t d2d3 = (size_t)d2 * (size_t)d3;
    size_t l = chunk_left(blockIdx.x, gridDim.x, d2d3);
    size_t r = chunk_right(blockIdx.x, gridDim.x, d2d3);
    for (size_t lv = l; lv < r; lv += batch_size) {
        batch_size = min(batch_size, r - lv);
        size_t offset = lv * (size_t)d1;
        __syncthreads();
        for (size_t idx = threadIdx.x; idx < batch_size * d1; idx += blockDim.x) {
			int u = (idx / d1);
            size_t i = (lv + u) % d2;
            size_t j = idx % d1;
            fn.set_i(i);
            smem[fn(j) + u * d1] = data[offset + idx];
        }
        
        __syncthreads();
        for (size_t idx = threadIdx.x; idx < batch_size * d1; idx += blockDim.x) {
            data[offset + idx] = smem[idx];
        }
    }
}

template<typename F, typename K, typename T>
void compress_row_launch(F fn, K kernel, T* data, int d1, int d2, int d3) {
	PRINT("Smem Compress %s\n", fn.getName().c_str());
	size_t smem_lim = shared_mem_per_block();
	size_t smem_size = smem_lim / 32;
	int n_threads = max_n_threads_per_sm() / 32;
	PRINT("\t# threads = %d\n", n_threads);
	int n_blocks = min(d2 * d3, get_num_block(kernel, n_threads, smem_size));
	PRINT("\t# blocks = %d\n", n_blocks);
	size_t batch_size = smem_size / (sizeof(T) * (size_t)d1);
	PRINT("\tbatch size = %zu\n", batch_size);
	kernel<<<n_blocks, n_threads, smem_size>>>(fn, data, batch_size, d1, d2, d3);
}

template<typename F, typename T>
__global__ void smem_row_gather_op(F fn, T* data, int d1, int d2, int d3) {
    T* smem = shared_memory<T>();
    
	for (size_t k = 0; k < d3; k++) {
		size_t kd1d2 = k * d1 * d2;
		for (size_t i = blockIdx.x; i < d2; i += gridDim.x) {
			fn.set_i(i);
			__syncthreads();
			for(size_t j = threadIdx.x; j < d1; j += blockDim.x) {
				smem[j] = data[j + i * d1 + kd1d2];
			}
			__syncthreads();
			for(size_t j = threadIdx.x; j < d1; j += blockDim.x) {
				data[j + i * d1 + kd1d2] = smem[fn(j)];
			}
		}
	}
}

template<typename F, typename T>
__global__ void smem_row_scatter_op(F fn, T* data, int d1, int d2, int d3) {
    T* smem = shared_memory<T>();
    
	for (size_t k = 0; k < d3; k++) {
		size_t kd1d2 = k * d1 * d2;
		for (size_t i = blockIdx.x; i < d2; i += gridDim.x) {
			fn.set_i(i);
			__syncthreads();
			for(size_t j = threadIdx.x; j < d1; j += blockDim.x) {
				smem[fn(j)] = data[j + i * d1 + kd1d2];
			}
			__syncthreads();
			for(size_t j = threadIdx.x; j < d1; j += blockDim.x) {
				data[j + i * d1 + kd1d2] = smem[j];
			}
		}
	}
}

template<typename F, typename K, typename T>
void smem_row_launch(F fn, K kernel, T* data, int d1, int d2, int d3) {
	PRINT("Smem %s\n", fn.getName().c_str());
	size_t smem_size = sizeof(T) * (size_t)d1;
	int n_threads = get_num_thread(d1);
	PRINT("\t# threads = %d\n", n_threads);
	int n_blocks = min(d2, get_num_block(kernel, n_threads, smem_size));
	kernel<<<n_blocks, n_threads, smem_size>>>(fn, data, d1, d2, d3);
}

template<typename F, typename T>
__global__ void gmem_multi_row_gather_op(F fn, T* data, T* tmp, int d1, int d2, int d3) {
	size_t offset = blockIdx.x * d1;
	for (size_t k = 0; k < d3; k++) {
		size_t kd1d2 = k * d1 * d2;
		for (size_t i = blockIdx.x; i < d2; i += gridDim.x) {
			fn.set_i(i);
			for (size_t j = threadIdx.x; j < d1; j += blockDim.x) {
				tmp[offset + j] = data[fn(j) + i * d1 + kd1d2];
			}
			__syncthreads();
			for (size_t j = threadIdx.x; j < d1; j += blockDim.x) {
				data[j + i * d1 + kd1d2] = tmp[offset + j];
			}
		}
	}
}

template<typename F, typename T>
__global__ void gmem_multi_row_scatter_op(F fn, T* data, T* tmp, int d1, int d2, int d3) {
	size_t offset = blockIdx.x * d1;
	for (size_t k = 0; k < d3; k++) {
		size_t kd1d2 = k * d1 * d2;
		for (size_t i = blockIdx.x; i < d2; i += gridDim.x) {
			fn.set_i(i);
			for (size_t j = threadIdx.x; j < d1; j += blockDim.x) {
				tmp[offset + fn(j)] = data[j + i * d1 + kd1d2];
			}
			__syncthreads();
			for (size_t j = threadIdx.x; j < d1; j += blockDim.x) {
				data[j + i * d1 + kd1d2] = tmp[offset + j];
			}
		}
	}
}

template<typename F, typename K, typename T>
void gmem_multi_row_launch(F fn, K kernel, T* data, int d1, int d2, int d3) {
	PRINT("Gmem Multi %s\n", fn.getName().c_str());
	
	int n_threads = 1024;
	int n_blocks = get_num_block(kernel, n_threads, 0);
	PRINT("\t# blocks = %d\n", n_blocks);
	
	T* tmp;
	size_t tmp_size = sizeof(T) * d1 * n_blocks;
	CudaSafeCall( hipMallocManaged(&tmp, tmp_size) );
	prefetch(tmp, tmp_size);

	void *kernelArgs[] = {
		(void *)&fn, (void *)&data, (void *)&tmp, (void *)&d1, (void *)&d2, (void *)&d3
	};
	CudaSafeCall( hipLaunchCooperativeKernel((void *)kernel,
										  n_blocks, n_threads, kernelArgs) );
	CudaSafeCall( hipFree(tmp) );
}

template<typename F, typename T>
__global__ void gmem_row_gather_op(F fn, T* data, T* tmp, int d1, int d2, int d3) {
    namespace cg = cooperative_groups;
    cg::grid_group g = cg::this_grid();

    size_t global_id = threadIdx.x + blockIdx.x * blockDim.x;
    size_t grid_size = gridDim.x * blockDim.x;
	for (size_t k = 0; k < d3; k++) {
		size_t kd1d2 = k * d1 * d2;
		for (size_t i = 0; i < d2; i++) {
			fn.set_i(i);
			g.sync();
			for (size_t j = global_id; j < d1; j += grid_size) {
				tmp[j] = data[fn(j) + i * d1 + kd1d2];
			}
			g.sync();
			for (size_t j = global_id; j < d1; j += grid_size) {
				data[j + i * d1 + kd1d2] = tmp[j];
			}
		}
	}
}

template<typename F, typename T>
__global__ void gmem_row_scatter_op(F fn, T* data, T* tmp, int d1, int d2, int d3) {
    namespace cg = cooperative_groups;
    cg::grid_group g = cg::this_grid();

    size_t global_id = threadIdx.x + blockIdx.x * blockDim.x;
    size_t grid_size = gridDim.x * blockDim.x;
	for (size_t k = 0; k < d3; k++) {
		size_t kd1d2 = k * d1 * d2;
		for (size_t i = 0; i < d2; i++) {
			fn.set_i(i);
			g.sync();
			for (size_t j = global_id; j < d1; j += grid_size) {
				//tmp[j] = d[rm(i, s(j))];
				tmp[fn(j)] = data[j + i * d1 + kd1d2];
			}
			g.sync();
			for (size_t j = global_id; j < d1; j += grid_size) {
				//d[rm(i, j)] = tmp[j];
				data[j + i * d1 + kd1d2] = tmp[j];
			}
		}
	}
}

template<typename F, typename K, typename T>
void gmem_row_launch(F fn, K kernel, T* data, int d1, int d2, int d3) {
	PRINT("Gmem %s\n", fn.getName().c_str());
	T* tmp;
	size_t tmp_size = sizeof(T) * d1;
	CudaSafeCall( hipMallocManaged(&tmp, tmp_size) );
	prefetch(tmp, tmp_size);
	int n_threads = 1024;
	int n_blocks = get_num_block(kernel, n_threads, 0);
	PRINT("\t# blocks = %d\n", n_blocks);
	void *kernelArgs[] = {
		(void *)&fn, (void *)&data, (void *)&tmp, (void *)&d1, (void *)&d2, (void *)&d3
	};
	CudaSafeCall( hipLaunchCooperativeKernel((void *)kernel,
										  n_blocks, n_threads, kernelArgs) );
	CudaSafeCall( hipFree(tmp) );
}

template<typename F, typename T>
void row_gather_op(F fn, T* data, int d1, int d2, int d3) {
	size_t smem_lim = shared_mem_per_block();
	if (2 * d1 * sizeof(T) <= smem_lim / 32) {
		compress_row_launch(fn, compress_row_gather_op<F, T>, data, d1, d2, d3);
    }
    else if (sizeof(T) * (size_t)d1 <= smem_lim) {
		smem_row_launch(fn, smem_row_gather_op<F, T>, data, d1, d2, d3);
    }
	else if (d1 * 64 / ((double)d1 * d2 * d3) < 0.1) {
		gmem_multi_row_launch(fn, gmem_multi_row_gather_op<F, T>, data, d1, d2, d3);
	}
	else {
        gmem_row_launch(fn, gmem_row_gather_op<F, T>, data, d1, d2, d3);
    }
}

template<typename F, typename T>
void row_scatter_op(F fn, T* data, int d1, int d2, int d3) {
	size_t smem_lim = shared_mem_per_block();
	if (2 * d1 * sizeof(T) <= smem_lim / 32) {
		compress_row_launch(fn, compress_row_scatter_op<F, T>, data, d1, d2, d3);
    }
    else if (sizeof(T) * (size_t)d1 <= smem_lim) {
		smem_row_launch(fn, smem_row_scatter_op<F, T>, data, d1, d2, d3);
    }
	else if (d1 * 64 / ((double)d1 * d2 * d3) < 0.1) {
		gmem_multi_row_launch(fn, gmem_multi_row_scatter_op<F, T>, data, d1, d2, d3);
	}
	else {
        gmem_row_launch(fn, gmem_row_scatter_op<F, T>, data, d1, d2, d3);
    }
}

template void row_gather_op(_2d::c2r::row_shuffle, float*, int, int, int);
template void row_gather_op(_2d::c2r::row_shuffle, double*, int, int, int);

template void row_gather_op(_2d::r2c::row_shuffle, float*, int, int, int);
template void row_gather_op(_2d::r2c::row_shuffle, double*, int, int, int);

template void row_scatter_op(_2d::r2c::row_scatter_shuffle, float*, int, int, int);
template void row_scatter_op(_2d::r2c::row_scatter_shuffle, double*, int, int, int);

}
}

namespace _2d {

template<typename F, typename T>
__global__ void compress_row_gather_op(F fn, T* data, size_t batch_size, int d1, int d2) {
    T* smem = shared_memory<T>();

    size_t l = chunk_left(blockIdx.x, gridDim.x, d2);
    size_t r = chunk_right(blockIdx.x, gridDim.x, d2);
    for (size_t lv = l; lv < r; lv += batch_size) {
        batch_size = min(batch_size, r - lv);
        size_t offset = lv * (size_t)d1;
        __syncthreads();
        for (size_t idx = threadIdx.x; idx < batch_size * d1; idx += blockDim.x) {
            smem[idx] = data[offset + idx];
        }
        
        __syncthreads();
        for (size_t idx = threadIdx.x; idx < batch_size * d1; idx += blockDim.x) {
            int u = (idx / d1);
            size_t i = (lv + u) % d2;
            size_t j = idx % d1;
            fn.set_i(i);
            data[offset + idx] = smem[fn(j) + u * d1];
        }
    }
}

template<typename F, typename T>
__global__ void compress_row_scatter_op(F fn, T* data, size_t batch_size, int d1, int d2) {
    T* smem = shared_memory<T>();

    size_t l = chunk_left(blockIdx.x, gridDim.x, d2);
    size_t r = chunk_right(blockIdx.x, gridDim.x, d2);
    for (size_t lv = l; lv < r; lv += batch_size) {
        batch_size = min(batch_size, r - lv);
        size_t offset = lv * (size_t)d1;
        __syncthreads();
        for (size_t idx = threadIdx.x; idx < batch_size * d1; idx += blockDim.x) {
			int u = (idx / d1);
            size_t i = (lv + u) % d2;
            size_t j = idx % d1;
            fn.set_i(i);
            smem[fn(j) + u * d1] = data[offset + idx];
        }
        
        __syncthreads();
        for (size_t idx = threadIdx.x; idx < batch_size * d1; idx += blockDim.x) {
            data[offset + idx] = smem[idx];
        }
    }
}

template<typename F, typename K, typename T>
void compress_row_launch(F fn, K kernel, T* data, int d1, int d2) {
	PRINT("Smem Compress %s\n", fn.getName().c_str());
	PRINT("\t(d1, d2) = (%d, %d)\n", d1, d2);
	size_t smem_lim = shared_mem_per_block();
	size_t smem_size = smem_lim / 32;
	int n_threads = max_n_threads_per_sm() / 32;
	PRINT("\t# threads = %d\n", n_threads);
	int n_blocks = min(d2, get_num_block(kernel, n_threads, smem_size));
	PRINT("\t# blocks = %d\n", n_blocks);
	size_t batch_size = smem_size / (sizeof(T) * (size_t)d1);
	PRINT("\tbatch size = %zu\n", batch_size);
	kernel<<<n_blocks, n_threads, smem_size>>>(fn, data, batch_size, d1, d2);
}

template<typename F, typename T>
__global__ void smem_row_gather_op(F fn, T* data, int d1, int d2) {
    T* smem = shared_memory<T>();
    
    for (size_t i = blockIdx.x; i < d2; i += gridDim.x) {
        fn.set_i(i);
        __syncthreads();
        for(size_t j = threadIdx.x; j < d1; j += blockDim.x) {
			smem[j] = data[j + i * d1];
        }
        __syncthreads();
        for(size_t j = threadIdx.x; j < d1; j += blockDim.x) {
			data[j + i * d1] = smem[fn(j)];
        }
    }
}

template<typename F, typename T>
__global__ void smem_row_scatter_op(F fn, T* data, int d1, int d2) {
    T* smem = shared_memory<T>();
    
    for (size_t i = blockIdx.x; i < d2; i += gridDim.x) {
        fn.set_i(i);
        __syncthreads();
        for(size_t j = threadIdx.x; j < d1; j += blockDim.x) {
			smem[fn(j)] = data[j + i * d1];
        }
        __syncthreads();
        for(size_t j = threadIdx.x; j < d1; j += blockDim.x) {
			data[j + i * d1] = smem[j];
        }
    }
}

template<typename F, typename K, typename T>
void smem_row_launch(F fn, K kernel, T* data, int d1, int d2) {
	PRINT("Smem %s\n", fn.getName().c_str());
	size_t smem_size = sizeof(T) * (size_t)d1;
	int n_threads = get_num_thread(d1);
	PRINT("\t# threads = %d\n", n_threads);
	int n_blocks = min(d2, get_num_block(kernel, n_threads, smem_size));
	kernel<<<n_blocks, n_threads, smem_size>>>(fn, data, d1, d2);
}

template<typename F, typename T>
__global__ void gmem_row_gather_op(F fn, T* data, T* tmp, int d1, int d2) {
    namespace cg = cooperative_groups;
    cg::grid_group g = cg::this_grid();

    size_t global_id = threadIdx.x + blockIdx.x * blockDim.x;
    size_t grid_size = gridDim.x * blockDim.x;
	for (size_t i = 0; i < d2; i++) {
		fn.set_i(i);
		g.sync();
		for (size_t j = global_id; j < d1; j += grid_size) {
			tmp[j] = data[fn(j) + i * d1];
		}
		g.sync();
		for (size_t j = global_id; j < d1; j += grid_size) {
			data[j + i * d1] = tmp[j];
		}
	}
}

template<typename F, typename T>
__global__ void gmem_row_scatter_op(F fn, T* data, T* tmp, int d1, int d2) {
    namespace cg = cooperative_groups;
    cg::grid_group g = cg::this_grid();

    size_t global_id = threadIdx.x + blockIdx.x * blockDim.x;
    size_t grid_size = gridDim.x * blockDim.x;
	for (size_t i = 0; i < d2; i++) {
		fn.set_i(i);
		g.sync();
		for (size_t j = global_id; j < d1; j += grid_size) {
			tmp[fn(j)] = data[j + i * d1];
		}
		g.sync();
		for (size_t j = global_id; j < d1; j += grid_size) {
			data[j + i * d1] = tmp[j];
		}
	}
}

template<typename F, typename K, typename T>
void gmem_row_launch(F fn, K kernel, T* data, int d1, int d2) {
	PRINT("Gmem %s\n", fn.getName().c_str());
	T* tmp;
	size_t tmp_size = sizeof(T) * d1;
	CudaSafeCall( hipMallocManaged(&tmp, tmp_size) );
	prefetch(tmp, tmp_size);
	int n_threads = 1024;
	int n_blocks = get_num_block(kernel, n_threads, 0) / 2;
	PRINT("\t# blocks = %d\n", n_blocks);
	void *kernelArgs[] = {
		(void *)&fn, (void *)&data, (void *)&tmp, (void *)&d1, (void *)&d2
	};
	CudaSafeCall( hipLaunchCooperativeKernel((void *)kernel,
										  n_blocks, n_threads, kernelArgs) );
	CudaSafeCall( hipFree(tmp) );
}

template<typename F, typename T>
void row_gather_op(F fn, T* data, int d1, int d2) {
	size_t smem_lim = shared_mem_per_block();
	if (2 * d1 * sizeof(T) <= smem_lim / 32) {
		compress_row_launch(fn, compress_row_gather_op<F, T>, data, d1, d2);
    }
    else if (sizeof(T) * (size_t)d1 <= smem_lim) {
		smem_row_launch(fn, smem_row_gather_op<F, T>, data, d1, d2);
    }
	else {
        gmem_row_launch(fn, gmem_row_gather_op<F, T>, data, d1, d2);
    }
}

template<typename F, typename T>
void row_scatter_op(F fn, T* data, int d1, int d2) {
	size_t smem_lim = shared_mem_per_block();
	if (2 * d1 * sizeof(T) <= smem_lim / 32) {
		compress_row_launch(fn, compress_row_scatter_op<F, T>, data, d1, d2);
    }
    else if (sizeof(T) * (size_t)d1 <= smem_lim) {
		smem_row_launch(fn, smem_row_scatter_op<F, T>, data, d1, d2);
    }
	else {
        gmem_row_launch(fn, gmem_row_scatter_op<F, T>, data, d1, d2);
    }
}

template void row_gather_op(c2r::row_shuffle, float*, int, int);
template void row_gather_op(c2r::row_shuffle, double*, int, int);

template void row_gather_op(r2c::row_shuffle, float*, int, int);
template void row_gather_op(r2c::row_shuffle, double*, int, int);

template void row_gather_op(_3d::_213::row_shuffle, float*, int, int);
template void row_gather_op(_3d::_213::row_shuffle, double*, int, int);

template void row_scatter_op(r2c::row_scatter_shuffle, float*, int, int);
template void row_scatter_op(r2c::row_scatter_shuffle, double*, int, int);


}

}
